// Copyright (c) 2021-2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include "dali/operators/numba_function/numba_func.h"

namespace dali {

template <typename GPUBackend>
NumbaFuncImpl<GPUBackend>::NumbaFuncImpl(const OpSpec &spec) : Base(spec) {
  // run_fn_cuda_ = spec.GetArgument<void*>("run_fn_cuda");
  run_fn_ = spec.GetArgument<uint64_t>("run_fn");
  setup_fn_ = spec.GetArgument<uint64_t>("setup_fn");
  batch_processing_ = spec.GetArgument<bool>("batch_processing");

  out_types_ = spec.GetRepeatedArgument<DALIDataType>("out_types");
  DALI_ENFORCE(out_types_.size() <= 6,
    make_string("Trying to specify ", out_types_.size(), " outputs. "
    "This operator can have at most 6 outputs."));
  in_types_ = spec.GetRepeatedArgument<DALIDataType>("in_types");
  DALI_ENFORCE(in_types_.size() <= 6,
    make_string("Trying to specify ", in_types_.size(), " inputs. "
      "This operator can have at most 6 inputs."));

  outs_ndim_ = spec.GetRepeatedArgument<int>("outs_ndim");
  DALI_ENFORCE(outs_ndim_.size() == out_types_.size(), make_string("Size of `outs_ndim` "
    "should match size of `out_types`."));
  for (size_t i = 0; i < outs_ndim_.size(); i++) {
    DALI_ENFORCE(outs_ndim_[i] >= 0, make_string(
      "All dimensions should be non negative. Value specified in `outs_ndim` at index ",
        i, " is negative."));
  }
  if (!setup_fn_) {
    DALI_ENFORCE(out_types_.size() == in_types_.size(),
      "Size of `out_types` should match size of `in_types` when `setup_fn` isn't provided.");
  }

  ins_ndim_ = spec.GetRepeatedArgument<int>("ins_ndim");
  DALI_ENFORCE(ins_ndim_.size() == in_types_.size(), make_string(
    "Size of `ins_dnim` should match size of `in_types`."));
  for (size_t i = 0; i < ins_ndim_.size(); i++) {
    DALI_ENFORCE(ins_ndim_[i] >= 0, make_string(
      "All dimensions should be non negative. Value specified in "
      "`ins_ndim` at index ", i, " is negative."));
  }
}

template <>
bool NumbaFuncImpl<GPUBackend>::SetupImpl(std::vector<OutputDesc> &output_desc,
    const workspace_t<GPUBackend> &ws) {
  int ninputs = ws.NumInput();
  int noutputs = out_types_.size();
  DALI_ENFORCE(in_types_.size() == static_cast<size_t>(ninputs), make_string(
    "Expected ", in_types_.size(), " inputs (basing on `in_types`), but got ", ninputs));
  DALI_ENFORCE(ins_ndim_.size() == static_cast<size_t>(ninputs), make_string(
    "Expected ", ins_ndim_.size(), " inputs (basing on `ins_ndim`), but got ", ninputs));

  output_desc.resize(out_types_.size());
  in_shapes_.resize(ninputs);
  for (int in_id = 0; in_id < ninputs; in_id++) {
    auto& in = ws.Input<GPUBackend>(in_id);
    in_shapes_[in_id] = in.shape();
    DALI_ENFORCE(in_shapes_[in_id].sample_dim() == ins_ndim_[in_id], make_string(
      "Number of dimensions passed in `ins_ndim` at index ", in_id,
      " doesn't match the number of dimensions of the input data: ",
      in_shapes_[in_id].sample_dim(), " != ", ins_ndim_[in_id]));
    DALI_ENFORCE(in.type() == in_types_[in_id], make_string(
      "Data type passed in `in_types` at index ", in_id, " doesn't match type of the input data: ",
      in.type(), " != ", in_types_[in_id]));
  }
  auto N = in_shapes_[0].num_samples();
  input_shape_ptrs_.resize(N * ninputs);
  for (int in_id = 0; in_id < ninputs; in_id++) {
    for (int i = 0; i < N; i++) {
      input_shape_ptrs_[N * in_id + i] =
        reinterpret_cast<uint64_t>(in_shapes_[in_id].tensor_shape_span(i).data());
    }
  }

  for (int i = 0; i < noutputs; i++) {
    const auto &in = ws.Input<GPUBackend>(i);
    output_desc[i] = {in.shape(), in.type()};
  }
  return true;
}

template <>
void NumbaFuncImpl<GPUBackend>::RunImpl(workspace_t<GPUBackend> &ws) {
  auto N = ws.Input<GPUBackend>(0).shape().num_samples();

  std::vector<uint64_t> out_ptrs;
  std::vector<uint64_t> in_ptrs;
  out_ptrs.resize(N * out_types_.size());
  in_ptrs.resize(N * in_types_.size());
  for (size_t out_id = 0; out_id < out_types_.size(); out_id++) {
    auto& out = ws.Output<GPUBackend>(out_id);
    for (int i = 0; i < N; i++) {
      out_ptrs[N * out_id + i] = reinterpret_cast<uint64_t>(out.raw_mutable_tensor(i));
    }
  }
  for (size_t in_id = 0; in_id < in_types_.size(); in_id++) {
    auto& in = ws.Input<GPUBackend>(in_id);
    for (int i = 0; i < N; i++) {
      in_ptrs[N * in_id + i] = reinterpret_cast<uint64_t>(in.raw_tensor(i));
    }
  }

  // const auto &input_ref = ws.template Input<GPUBackend>(0);
  // auto &output_ref = ws.template Output<GPUBackend>(0);
  // output_ref.SetLayout(input_ref.GetLayout());
  // auto input = view<const float, 2>(input_ref);
  // auto output = view<float, 2>(output_ref);
  // void* p_v = static_cast<void*>(out_ptrs.data());
  std::vector<void*> out_ptrs_v {NULL};
  void** data = NULL;
  dim3 block(10, 10);
  dim3 grid(1, 1);
  // auto fun = *reinterpret_cast<void**>(run_fn_);
  hipError_t result = hipLaunchKernel(reinterpret_cast<void*>(run_fn_), grid, block, data, 0, ws.stream());
  // hipError_t result = hipLaunchKernel(reinterpret_cast<const void*>(run_fn_cuda_), grid, block, data, 0, ws.stream());
  printf("Result: %d \n", result);
  // hipError_t result = hipModuleLaunchKernel(static_cast<hipFunction_t>(run_fn_), 1, 1, 1, 1, 1, 1, 0, 0, (void**)(out_ptrs_v.data()), NULL);
  // if (result != 0) {
    // const char *msg = hipGetErrorString(result);
    // printf("error: %s failed with error %s\n", result, msg);
  // }
}

DALI_REGISTER_OPERATOR(NumbaFuncImpl, NumbaFuncImpl<GPUBackend>, GPU);

}  // namespace dali

